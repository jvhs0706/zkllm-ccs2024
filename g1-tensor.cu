#include "hip/hip_runtime.h"

#include <iostream>
#include <iomanip>
#include "fr-tensor.cuh"
#include "g1-tensor.cuh"
#include "ioutils.cuh"

using namespace std;


DEVICE Fp_t Fp_minus(Fp_t a) {
	return blstrs__fp__Fp_sub(blstrs__fp__Fp_ZERO, a);
}

DEVICE G1Affine_t G1Affine_minus(G1Affine_t a) {
	return {a.x, Fp_minus(a.y)};
}

DEVICE G1Jacobian_t G1Jacobian_minus(G1Jacobian_t a) {
	return {a.x, Fp_minus(a.y), a.z};
}

ostream& operator<<(ostream& os, const Fp_t& x)
{
  os << "0x" << std::hex;
  for (uint i = 12; i > 0; -- i)
  {
    os << std::setfill('0') << std::setw(8) << x.val[i - 1];
  }
  return os << std::dec << std::setw(0) << std::setfill(' ');
}

ostream& operator<<(ostream& os, const G1Affine_t& g)
{
	os << "(" << g.x << ", " << g.y << ")";
	return os;
}

ostream& operator<<(ostream& os, const G1Jacobian_t& g)
{
	os << "(" << g.x << ", " << g.y <<  ", " << g.z << ")";
	return os;
}


G1Tensor::G1Tensor(uint size): size(size) {}

// Implement G1Affine

G1TensorAffine::G1TensorAffine(const G1TensorAffine& t): G1Tensor(t.size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Affine_t) * size);
    hipMemcpy(gpu_data, t.gpu_data, sizeof(G1Affine_t) * size, hipMemcpyDeviceToDevice);
}

G1TensorAffine::G1TensorAffine(uint size): G1Tensor(size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Affine_t) * size);
}

KERNEL void G1Affine_assign_broadcast(GLOBAL G1Affine_t* arr, GLOBAL G1Affine_t g, uint n)
{
	const uint gid = GET_GLOBAL_ID();
	if (gid >= n) return;
	arr[gid] = g;
}

G1TensorAffine::G1TensorAffine(uint size, const G1Affine_t& g): G1Tensor(size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Affine_t) * size);
    G1Affine_assign_broadcast<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, g, size);
    hipDeviceSynchronize();
}

G1TensorAffine::G1TensorAffine(uint size, const G1Affine_t* cpu_data): G1Tensor(size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Affine_t) * size);
    hipMemcpy(gpu_data, cpu_data, sizeof(G1Affine_t) * size, hipMemcpyHostToDevice);
}

G1TensorAffine::~G1TensorAffine()
{
    hipFree(gpu_data);
    gpu_data = nullptr;
}

void G1TensorAffine::save(const string& filename) const
{
    savebin(filename, gpu_data, size * sizeof(G1Affine_t));
}

G1TensorAffine::G1TensorAffine(const string& filename): G1Tensor(findsize(filename) / sizeof(G1Affine_t)), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, size * sizeof(G1Affine_t));
    loadbin(filename, gpu_data, size * sizeof(G1Affine_t));
}

G1Affine_t G1TensorAffine::operator()(uint idx) const
{
    G1Affine_t out;
    hipMemcpy(&out, gpu_data + idx, sizeof(G1Affine_t), hipMemcpyDeviceToHost);
    return out;
}

KERNEL void G1_affine_elementwise_minus(GLOBAL G1Affine_t* arr_in, GLOBAL G1Affine_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = {arr_in[gid].x, blstrs__fp__Fp_sub(blstrs__fp__Fp_ZERO, arr_in[gid].y)};
}

G1TensorAffine G1TensorAffine::operator-() const
{
    G1TensorAffine out(size);
    G1_affine_elementwise_minus<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, out.gpu_data, size);
    hipDeviceSynchronize();
    return out;
}


// Implement G1TensorJacobian

G1TensorJacobian::G1TensorJacobian(const G1TensorJacobian& t): G1Tensor(t.size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Jacobian_t) * size);
    hipMemcpy(gpu_data, t.gpu_data, sizeof(G1Jacobian_t) * size, hipMemcpyDeviceToDevice);
}

G1TensorJacobian::G1TensorJacobian(uint size): G1Tensor(size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Jacobian_t) * size);
}

G1TensorJacobian::G1TensorJacobian(uint size, const G1Jacobian_t* cpu_data): G1Tensor(size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Jacobian_t) * size);
    hipMemcpy(gpu_data, cpu_data, sizeof(G1Jacobian_t) * size, hipMemcpyHostToDevice);
}

KERNEL void G1Jacobian_assign_broadcast(GLOBAL G1Jacobian_t* arr, G1Jacobian_t g, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr[gid] = g;
}

G1TensorJacobian::G1TensorJacobian(uint size, const G1Jacobian_t& g): G1Tensor(size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Jacobian_t) * size);
    G1Jacobian_assign_broadcast<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, g, size);
    hipDeviceSynchronize();
}

KERNEL void G1_affine_to_jacobian(GLOBAL G1Affine_t* arr_affine, GLOBAL G1Jacobian_t* arr_jacobian, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_jacobian[gid] = {arr_affine[gid].x, arr_affine[gid].y, blstrs__fp__Fp_ONE};
}

G1TensorJacobian::G1TensorJacobian(const G1TensorAffine& affine_tensor): G1Tensor(affine_tensor.size), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, sizeof(G1Jacobian_t) * size);
    G1_affine_to_jacobian<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(affine_tensor.gpu_data, gpu_data, size);
    hipDeviceSynchronize();
}

G1TensorJacobian::~G1TensorJacobian()
{
    hipFree(gpu_data);
    gpu_data = nullptr;
}

void G1TensorJacobian::save(const string& filename) const
{
    savebin(filename, gpu_data, size * sizeof(G1Jacobian_t));
}

G1TensorJacobian::G1TensorJacobian(const string& filename): G1Tensor(findsize(filename) / sizeof(G1Jacobian_t)), gpu_data(nullptr)
{
    hipMalloc((void **)&gpu_data, size * sizeof(G1Jacobian_t));
    loadbin(filename, gpu_data, size * sizeof(G1Jacobian_t));
}

G1Jacobian_t G1TensorJacobian::operator()(uint idx) const
{
	G1Jacobian_t out;
	hipMemcpy(&out, gpu_data + idx, sizeof(G1Jacobian_t), hipMemcpyDeviceToHost);
	return out;
}

KERNEL void G1_jacobian_elementwise_minus(GLOBAL G1Jacobian_t* arr_in, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = {arr_in[gid].x, blstrs__fp__Fp_sub(blstrs__fp__Fp_ZERO, arr_in[gid].y), arr_in[gid].z};
}

G1TensorJacobian G1TensorJacobian::operator-() const
{
    G1TensorJacobian out(size);
    G1_jacobian_elementwise_minus<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, out.gpu_data, size);
    hipDeviceSynchronize();
    return out;
}

KERNEL void G1_jacobian_elementwise_add(GLOBAL G1Jacobian_t* arr1, GLOBAL G1Jacobian_t* arr2, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add(arr1[gid], arr2[gid]);
}

KERNEL void G1_jacobian_broadcast_add(GLOBAL G1Jacobian_t* arr, G1Jacobian_t x, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add(arr[gid], x);
}

KERNEL void G1_jacobian_elementwise_madd(GLOBAL G1Jacobian_t* arr1, GLOBAL G1Affine_t* arr2, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add_mixed(arr1[gid], arr2[gid]);
}

KERNEL void G1_jacobian_broadcast_madd(GLOBAL G1Jacobian_t* arr, G1Affine_t x, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add_mixed(arr[gid], x);
}

G1TensorJacobian G1TensorJacobian::operator+(const G1TensorJacobian& t) const
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1TensorJacobian out(size);
	G1_jacobian_elementwise_add<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}
    
G1TensorJacobian G1TensorJacobian::operator+(const G1TensorAffine& t) const
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1TensorJacobian out(size);
	G1_jacobian_elementwise_madd<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}

G1TensorJacobian G1TensorJacobian::operator+(const G1Jacobian_t& x) const
{
	G1TensorJacobian out(size);
	G1_jacobian_broadcast_add<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}

G1TensorJacobian G1TensorJacobian::operator+(const G1Affine_t& x) const
{
	G1TensorJacobian out(size);
	G1_jacobian_broadcast_madd<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}

G1TensorJacobian& G1TensorJacobian::operator+=(const G1TensorJacobian& t)
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1_jacobian_elementwise_add<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}
    
G1TensorJacobian& G1TensorJacobian::operator+=(const G1TensorAffine& t)
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1_jacobian_elementwise_madd<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}

G1TensorJacobian& G1TensorJacobian::operator+=(const G1Jacobian_t& x)
{
	G1_jacobian_broadcast_add<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}

G1TensorJacobian& G1TensorJacobian::operator+=(const G1Affine_t& x)
{
	G1_jacobian_broadcast_madd<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}

KERNEL void G1_jacobian_elementwise_sub(GLOBAL G1Jacobian_t* arr1, GLOBAL G1Jacobian_t* arr2, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add(arr1[gid], G1Jacobian_minus(arr2[gid]));
}

KERNEL void G1_jacobian_broadcast_sub(GLOBAL G1Jacobian_t* arr, G1Jacobian_t x, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add(arr[gid], G1Jacobian_minus(x));
}

KERNEL void G1_jacobian_elementwise_msub(GLOBAL G1Jacobian_t* arr1, GLOBAL G1Affine_t* arr2, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add_mixed(arr1[gid], G1Affine_minus(arr2[gid]));
}

KERNEL void G1_jacobian_broadcast_msub(GLOBAL G1Jacobian_t* arr, G1Affine_t x, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = blstrs__g1__G1Affine_add_mixed(arr[gid], G1Affine_minus(x));
}

G1TensorJacobian G1TensorJacobian::operator-(const G1TensorJacobian& t) const
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1TensorJacobian out(size);
	G1_jacobian_elementwise_sub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}
    
G1TensorJacobian G1TensorJacobian::operator-(const G1TensorAffine& t) const
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1TensorJacobian out(size);
	G1_jacobian_elementwise_msub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}

G1TensorJacobian G1TensorJacobian::operator-(const G1Jacobian_t& x) const
{
	G1TensorJacobian out(size);
	G1_jacobian_broadcast_sub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}

G1TensorJacobian G1TensorJacobian::operator-(const G1Affine_t& x) const
{
	G1TensorJacobian out(size);
	G1_jacobian_broadcast_msub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, out.gpu_data, size);
	hipDeviceSynchronize();
	return out;
}

G1TensorJacobian& G1TensorJacobian::operator-=(const G1TensorJacobian& t)
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1_jacobian_elementwise_sub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}
    
G1TensorJacobian& G1TensorJacobian::operator-=(const G1TensorAffine& t)
{
	if (size != t.size) throw std::runtime_error("Incompatible dimensions");
	G1_jacobian_elementwise_msub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}

G1TensorJacobian& G1TensorJacobian::operator-=(const G1Jacobian_t& x)
{
	G1_jacobian_broadcast_sub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}

G1TensorJacobian& G1TensorJacobian::operator-=(const G1Affine_t& x)
{
	G1_jacobian_broadcast_msub<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, x, gpu_data, size);
	hipDeviceSynchronize();
	return *this;
}

KERNEL void G1Jacobian_sum_reduction(GLOBAL G1Jacobian_t *arr, GLOBAL G1Jacobian_t *output, uint n) {
    extern __shared__ G1Jacobian_t g1sum_sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (2 * blockDim.x) + threadIdx.x;

    // Load input into shared memory
    g1sum_sdata[tid] = (i < n) ? arr[i] : blstrs__g1__G1Affine_ZERO;
    if (i + blockDim.x < n) g1sum_sdata[tid] = blstrs__g1__G1Affine_add(g1sum_sdata[tid], arr[i + blockDim.x]);

    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            g1sum_sdata[tid] = blstrs__g1__G1Affine_add(g1sum_sdata[tid], g1sum_sdata[tid + s]);
        }
        __syncthreads();
    }

    // Write the result for this block to output
    if (tid == 0) output[blockIdx.x] = g1sum_sdata[0];
}

G1Jacobian_t G1TensorJacobian::sum() const
{
    G1Jacobian_t *ptr_input, *ptr_output;
    uint curSize = size;
    hipMalloc((void**)&ptr_input, size * sizeof(G1Jacobian_t));
    hipMalloc((void**)&ptr_output, ((size + 1)/ 2) * sizeof(G1Jacobian_t));
    hipMemcpy(ptr_input, gpu_data, size * sizeof(G1Jacobian_t), hipMemcpyDeviceToDevice);

    while(curSize > 1) {
        uint gridSize = (curSize + G1NumThread - 1) / G1NumThread;
        G1Jacobian_sum_reduction<<<gridSize, G1NumThread, G1JacobianSharedMemorySize>>>(ptr_input, ptr_output, curSize);
        hipDeviceSynchronize(); // Ensure kernel completion before proceeding
        
        // Swap pointers. Use the output from this step as the input for the next step.
        G1Jacobian_t *temp = ptr_input;
        ptr_input = ptr_output;
        ptr_output = temp;
        
        curSize = gridSize;  // The output size is equivalent to the grid size used in the kernel launch
    }

    G1Jacobian_t finalSum;
    hipMemcpy(&finalSum, ptr_input, sizeof(G1Jacobian_t), hipMemcpyDeviceToHost);

    hipFree(ptr_input);
    hipFree(ptr_output);

    return finalSum;
}

DEVICE G1Jacobian_t G1Jacobian_mul(G1Jacobian_t a, Fr_t x) {
    G1Jacobian_t out = blstrs__g1__G1Affine_ZERO;
    #pragma unroll
    for (uint i = 0; i < 256; ++i) {
        if ((x.val[i / 32] >> (i % 32)) & 1U) out = blstrs__g1__G1Affine_add(out, a); // the i-th bit of x
        a = blstrs__g1__G1Affine_double(a); // (1 << i) * original_a
    }
    return out;
}


KERNEL void G1_jacobian_elementwise_mul(GLOBAL G1Jacobian_t* arr_g1, GLOBAL Fr_t* arr_fr, GLOBAL G1Jacobian_t* arr_out, uint n)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= n) return;
    arr_out[gid] = G1Jacobian_mul(arr_g1[gid], arr_fr[gid]);
}

KERNEL void G1_jacobian_elementwise_mul_broadcast(GLOBAL G1Jacobian_t* arr_g1, GLOBAL Fr_t* arr_fr, GLOBAL G1Jacobian_t* arr_out, uint n, uint m)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= m * n) return;
    arr_out[gid] = G1Jacobian_mul(arr_g1[gid % n], arr_fr[gid]);
}

G1TensorJacobian G1TensorJacobian::operator*(const FrTensor& scalar_tensor) const {
    if (scalar_tensor.size % size != 0) throw std::runtime_error("Incompatible dimensions");
    uint m = scalar_tensor.size / size;
    G1TensorJacobian out(scalar_tensor.size);  // output size will be same as scalar_tensor
    G1_jacobian_elementwise_mul_broadcast<<<(scalar_tensor.size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, scalar_tensor.gpu_data, out.gpu_data, size, m);
    hipDeviceSynchronize();
    return out;
}

G1TensorJacobian& G1TensorJacobian::operator*=(const FrTensor& scalar_tensor) {
    if (size != scalar_tensor.size) throw std::runtime_error("Incompatible dimensions 01");
    G1_jacobian_elementwise_mul<<<(size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, scalar_tensor.gpu_data, gpu_data, size);
    hipDeviceSynchronize();
    return *this;
}

KERNEL void G1_me_step(GLOBAL G1Jacobian_t *arr_in, GLOBAL G1Jacobian_t *arr_out, Fr_t x, uint in_size, uint out_size)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= out_size) return;

    Fr_t x_unmont = blstrs__scalar__Scalar_unmont(x);
    
    uint gid0 = 2 * gid;
    uint gid1 = 2 * gid + 1;
    if (gid1 < in_size) arr_out[gid] = blstrs__g1__G1Affine_add(arr_in[gid0], G1Jacobian_mul(blstrs__g1__G1Affine_add(arr_in[gid1], G1Jacobian_minus(arr_in[gid0])), x_unmont));
    else if (gid0 < in_size) arr_out[gid] = blstrs__g1__G1Affine_add(arr_in[gid0], G1Jacobian_minus(G1Jacobian_mul(arr_in[gid0], x_unmont)));
    else arr_out[gid] = blstrs__g1__G1Affine_ZERO;
}

G1Jacobian_t G1_me(const G1TensorJacobian& t, vector<Fr_t>::const_iterator begin, vector<Fr_t>::const_iterator end)
{
    G1TensorJacobian t_new((t.size + 1) / 2);
    if (begin >= end) return t(0);
    G1_me_step<<<(t_new.size+G1NumThread-1)/G1NumThread,G1NumThread>>>(t.gpu_data, t_new.gpu_data, *begin, t.size, t_new.size);
    hipDeviceSynchronize();
    return G1_me(t_new, begin + 1, end);
}

G1Jacobian_t G1TensorJacobian::operator()(const vector<Fr_t>& u) const
{
    uint log_dim = u.size();
    if (size <= (1 << (log_dim - 1)) || size > (1 << log_dim)) throw std::runtime_error("Incompatible dimensions");
    return G1_me(*this, u.begin(), u.end());
}

// OLD VERSION
// KERNEL void G1Jacobian_rowwise_sum_step(const G1Jacobian_t* arr_in, G1Jacobian_t* arr_out, uint nrow, uint ncol_in, uint ncol_out)
// {
//     auto gid = GET_GLOBAL_ID();
//     auto row_id = gid / ncol_out;
//     auto col_id = gid % ncol_out;

//     if (row_id < nrow && col_id < ncol_out) {
//         // need to consider the case when ncol_in is odd
//         if (2 * col_id + 1 == ncol_in) {
//             arr_out[row_id * ncol_out + col_id] = arr_in[row_id * ncol_in + 2 * col_id];
//         } else {
//             arr_out[row_id * ncol_out + col_id] = blstrs__g1__G1Affine_add(arr_in[row_id * ncol_in + 2 * col_id], arr_in[row_id * ncol_in + 2 * col_id + 1]);
//         }
//     }
// }

// blockdim.x == blockdim.y == TILE_WIDTH
// ncol_out == (ncol + blockdim.x - 1) / blockdim.x
KERNEL void G1Jacobian_rowwise_sum_reduction(const G1Jacobian_t* arr_in, G1Jacobian_t* arr_out, uint nrow, uint ncol, uint ncol_out)
{
    __shared__ G1Jacobian_t rwsum_data[TILE_WIDTH][TILE_WIDTH];

    auto row_id = blockIdx.y * TILE_WIDTH + threadIdx.y;
    auto row_tid = threadIdx.y;

    auto col_id = blockIdx.x * TILE_WIDTH + threadIdx.x;
    auto col_tid = threadIdx.x;

    // Load input into shared memory
    rwsum_data[row_tid][col_tid] = (row_id < nrow && col_id < ncol) ? arr_in[row_id * ncol + col_id] : blstrs__g1__G1Affine_ZERO;
    __syncthreads();

    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
        if (col_tid < s && col_id + s < ncol) {
            rwsum_data[row_tid][col_tid] = blstrs__g1__G1Affine_add(rwsum_data[row_tid][col_tid], rwsum_data[row_tid][col_tid + s]);
        }
        __syncthreads();
    }
    
    // Write the result for this block to output
    if (col_tid == 0 && row_id < nrow){
        arr_out[row_id * ncol_out + blockIdx.x] = rwsum_data[row_tid][0];
    }
}

// OLD VERSION
// G1TensorJacobian G1TensorJacobian::rowwise_sum(uint nrow, uint ncol) const
// {
//     if (size != nrow * ncol) throw std::runtime_error("Incompatible dimensions");

//     G1TensorJacobian out(nrow);

//     G1TensorJacobian temp0 (*this);
//     G1TensorJacobian temp1 (nrow * ((ncol + 1) / 2));
    
//     auto ptr0 = temp0.gpu_data;
//     auto ptr1 = temp1.gpu_data;

//     while (ncol > 1) {
//         auto ncol_out = (ncol + 1) >> 1;
//         G1Jacobian_rowwise_sum_step<<<(nrow * ncol_out + G1NumThread - 1) / G1NumThread, G1NumThread>>>(ptr0, ptr1, nrow, ncol, ncol_out);
//         hipDeviceSynchronize();
//         // swap ptr0, ptr1
//         auto temp_ptr = ptr0;
//         ptr0 = ptr1;
//         ptr1 = temp_ptr;
//         ncol = ncol_out;
//     }

//     hipMemcpy(out.gpu_data, ptr0, nrow * sizeof(G1Jacobian_t), hipMemcpyDeviceToDevice);
//     return out;
// }



G1TensorJacobian G1TensorJacobian::rowwise_sum(uint nrow, uint ncol) const
{
    if (size != nrow * ncol) throw std::runtime_error("Incompatible dimensions");
    G1Jacobian_t *ptr_input, *ptr_output;
    uint curNumCol = ncol;
    uint nextNumCol = (curNumCol + G1RowwiseSumTileWidth - 1) / G1RowwiseSumTileWidth;
    hipMalloc((void**)&ptr_input, nrow * curNumCol * sizeof(G1Jacobian_t));
    hipMalloc((void**)&ptr_output, nrow * nextNumCol * sizeof(G1Jacobian_t));
    hipMemcpy(ptr_input, gpu_data, nrow * ncol * sizeof(G1Jacobian_t), hipMemcpyDeviceToDevice);
    uint gridDimY = (nrow + G1RowwiseSumTileWidth - 1) / G1RowwiseSumTileWidth;

    while (curNumCol > 1)
    {
        // cout << "Current size = " << nrow << " x " << curNumCol << endl;
        uint gridDimX = (curNumCol + G1RowwiseSumTileWidth - 1) / G1RowwiseSumTileWidth;
        G1Jacobian_rowwise_sum_reduction<<<dim3(gridDimX, gridDimY), dim3(G1RowwiseSumTileWidth, G1RowwiseSumTileWidth)>>>(ptr_input, ptr_output, nrow, curNumCol, nextNumCol);
        hipDeviceSynchronize();
        // Swap pointers. Use the output from this step as the input for the next step.
        G1Jacobian_t *temp = ptr_input;
        ptr_input = ptr_output;
        ptr_output = temp;

        curNumCol = nextNumCol;
        nextNumCol = (curNumCol + G1RowwiseSumTileWidth - 1) / G1RowwiseSumTileWidth;
    }

    G1TensorJacobian out(nrow);
    hipMemcpy(out.gpu_data, ptr_input, nrow * sizeof(G1Jacobian_t), hipMemcpyDeviceToDevice);

    hipFree(ptr_input);
    hipFree(ptr_output);

    return out;
}