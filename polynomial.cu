#include "hip/hip_runtime.h"
#include "polynomial.cuh"

//kernel for operator+
__global__ void addKernel(const Fr_t* a, const Fr_t* b, Fr_t* c)
{
    *c = blstrs__scalar__Scalar_add(*a, *b);
}

Fr_t operator+(const Fr_t& a, const Fr_t& b)
{
    //copy a and b to cuda
    Fr_t* a_cuda, *b_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, sizeof(Fr_t));
    hipMalloc((void**)&b_cuda, sizeof(Fr_t));
    hipMalloc((void**)&c_cuda, sizeof(Fr_t));
    hipMemcpy(a_cuda, &a, sizeof(Fr_t), hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, &b, sizeof(Fr_t), hipMemcpyHostToDevice);
    addKernel<<<1, 1>>>(a_cuda, b_cuda, c_cuda);
    hipDeviceSynchronize();
    Fr_t c;
    hipMemcpy(&c, c_cuda, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return c;
}

//kernel for operator-
__global__ void subKernel(const Fr_t* a, const Fr_t* b, Fr_t* c)
{
    *c = blstrs__scalar__Scalar_sub(*a, *b);
}

Fr_t operator-(const Fr_t& a, const Fr_t& b)
{
    //copy a and b to cuda
    Fr_t* a_cuda, *b_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, sizeof(Fr_t));
    hipMalloc((void**)&b_cuda, sizeof(Fr_t));
    hipMalloc((void**)&c_cuda, sizeof(Fr_t));
    hipMemcpy(a_cuda, &a, sizeof(Fr_t), hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, &b, sizeof(Fr_t), hipMemcpyHostToDevice);
    subKernel<<<1, 1>>>(a_cuda, b_cuda, c_cuda);
    hipDeviceSynchronize();
    Fr_t c;
    hipMemcpy(&c, c_cuda, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return c;
}

__global__ void negKernel(const Fr_t* a, Fr_t* c)
{
    *c = blstrs__scalar__Scalar_sub({0, 0, 0, 0, 0, 0, 0, 0}, *a);
}

Fr_t operator-(const Fr_t& a)
{
    //copy a to cuda
    Fr_t* a_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, sizeof(Fr_t));
    hipMalloc((void**)&c_cuda, sizeof(Fr_t));
    hipMemcpy(a_cuda, &a, sizeof(Fr_t), hipMemcpyHostToDevice);
    negKernel<<<1, 1>>>(a_cuda, c_cuda);
    hipDeviceSynchronize();
    Fr_t c;
    hipMemcpy(&c, c_cuda, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(a_cuda);
    hipFree(c_cuda);
    return c;
}

__global__ void mulKernel(const Fr_t* a, const Fr_t* b, Fr_t* c)
{
    *c = blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(*a, *b));
}

Fr_t operator*(const Fr_t& a, const Fr_t& b)
{
    //copy a and b to cuda
    Fr_t* a_cuda, *b_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, sizeof(Fr_t));
    hipMalloc((void**)&b_cuda, sizeof(Fr_t));
    hipMalloc((void**)&c_cuda, sizeof(Fr_t));
    hipMemcpy(a_cuda, &a, sizeof(Fr_t), hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, &b, sizeof(Fr_t), hipMemcpyHostToDevice);
    mulKernel<<<1, 1>>>(a_cuda, b_cuda, c_cuda);
    hipDeviceSynchronize();
    Fr_t c;
    hipMemcpy(&c, c_cuda, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return c;
}

__global__ void divKernel(const Fr_t* a, const Fr_t* b, Fr_t* c)
{
    auto a_mont = blstrs__scalar__Scalar_mont(*a);
    auto b_mont = blstrs__scalar__Scalar_mont(*b);
    *c = blstrs__scalar__Scalar_unmont(blstrs__scalar__Scalar_div(a_mont, b_mont));
}

Fr_t operator/(const Fr_t& a, const Fr_t& b)
{
    if (!b.val[0] && !b.val[1] && !b.val[2] && !b.val[3] && !b.val[4] && !b.val[5] && !b.val[6] && !b.val[7]) {
        throw std::runtime_error("divide by zero");
    }
    //copy a and b to cuda
    Fr_t* a_cuda, *b_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, sizeof(Fr_t));
    hipMalloc((void**)&b_cuda, sizeof(Fr_t));
    hipMalloc((void**)&c_cuda, sizeof(Fr_t));
    hipMemcpy(a_cuda, &a, sizeof(Fr_t), hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, &b, sizeof(Fr_t), hipMemcpyHostToDevice);
    divKernel<<<1, 1>>>(a_cuda, b_cuda, c_cuda);
    hipDeviceSynchronize();
    Fr_t c;
    hipMemcpy(&c, c_cuda, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);
    return c;
}

__global__ void invKernel(const Fr_t* a, Fr_t* c)
{   
    Fr_t a_mont = blstrs__scalar__Scalar_mont(*a);
    *c = blstrs__scalar__Scalar_unmont(blstrs__scalar__Scalar_inverse(a_mont));
}

Fr_t inv(const Fr_t& a)
{   
    if (!a.val[0] && !a.val[1] && !a.val[2] && !a.val[3] && !a.val[4] && !a.val[5] && !a.val[6] && !a.val[7]) {
        throw std::runtime_error("divide by zero");
    }
    //copy a to cuda
    Fr_t* a_cuda, *c_cuda;
    hipMalloc((void**)&a_cuda, sizeof(Fr_t));
    hipMalloc((void**)&c_cuda, sizeof(Fr_t));
    hipMemcpy(a_cuda, &a, sizeof(Fr_t), hipMemcpyHostToDevice);
    invKernel<<<1, 1>>>(a_cuda, c_cuda);
    hipDeviceSynchronize();
    Fr_t c;
    hipMemcpy(&c, c_cuda, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(a_cuda);
    hipFree(c_cuda);
    return c;
}

Polynomial::Polynomial() : degree_(0), coefficients_(nullptr) {}

Polynomial::Polynomial(int degree) : degree_(degree) {
    hipMalloc((void**)&coefficients_, (degree + 1) * sizeof(Fr_t));
    hipMemset(coefficients_, 0, (degree + 1) * sizeof(Fr_t));
}

Polynomial::Polynomial(int degree, Fr_t* coefficients) : degree_(degree) {
    hipMalloc((void**)&coefficients_, (degree + 1) * sizeof(Fr_t));
    hipMemcpy(coefficients_, coefficients, (degree + 1) * sizeof(Fr_t), hipMemcpyHostToDevice);
}

Polynomial::Polynomial(const Polynomial& other) : degree_(other.degree_) {
    hipMalloc((void**)&coefficients_, (degree_ + 1) * sizeof(Fr_t));
    hipMemcpy(coefficients_, other.coefficients_, (degree_ + 1) * sizeof(Fr_t), hipMemcpyDeviceToDevice);
}

Polynomial::Polynomial(const Fr_t& constant) : degree_(0) {
    hipMalloc((void**)&coefficients_, sizeof(Fr_t));
    hipMemcpy(coefficients_, &constant, sizeof(Fr_t), hipMemcpyHostToDevice);
}

Polynomial::Polynomial(const vector<Fr_t>& coefficients) : degree_(coefficients.size() - 1) {
    hipMalloc((void**)&coefficients_, (degree_ + 1) * sizeof(Fr_t));
    hipMemcpy(coefficients_, coefficients.data(), (degree_ + 1) * sizeof(Fr_t), hipMemcpyHostToDevice);
}

Polynomial::~Polynomial() {
    if (coefficients_ != nullptr) {
        hipFree(coefficients_);
    }
}

__global__ void polyAddKernel(int n, int m, const Fr_t* a, const Fr_t* b, Fr_t* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (i < m) {
            c[i] = blstrs__scalar__Scalar_add(a[i], b[i]);
        } else {
            c[i] = a[i];
        }
    } else if (i < m) {
        c[i] = b[i];
    }
}

Polynomial Polynomial::operator+(const Polynomial& other) {
    int resultDegree = max(degree_, other.degree_);
    Polynomial result(resultDegree);

    polyAddKernel<<<1, resultDegree + 1>>>(degree_ + 1, other.degree_ + 1, coefficients_, other.coefficients_, result.coefficients_);
    hipDeviceSynchronize();

    return result;
}

__global__ void polySubKernel(int n, int m, const Fr_t* a, const Fr_t* b, Fr_t* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (i < m) {
            c[i] = blstrs__scalar__Scalar_sub(a[i], b[i]);
        } else {
            c[i] = a[i];
        }
    } else if (i < m) {
        c[i] = b[i];
    }
}

Polynomial Polynomial::operator-(const Polynomial& other) {
    int resultDegree = max(degree_, other.degree_);
    Polynomial result(resultDegree);

    polySubKernel<<<1, resultDegree + 1>>>(degree_ + 1, other.degree_ + 1, coefficients_, other.coefficients_, result.coefficients_);
    hipDeviceSynchronize();

    return result;
}

__global__ void polyMulKernel(int n, int m, const Fr_t* a, const Fr_t* b, Fr_t* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n + m - 1) {
        c[i] = {0, 0, 0, 0, 0, 0, 0, 0};
        for (int j = max(0, i - m + 1); j <= min(i, n - 1); j++) {
            c[i] = blstrs__scalar__Scalar_add(c[i], blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(a[j], b[i - j])));
        }
    }
}

Polynomial Polynomial::operator*(const Polynomial& other) {
    int resultDegree = degree_ + other.degree_;
    Polynomial result(resultDegree);

    polyMulKernel<<<1, resultDegree + 1>>>(degree_ + 1, other.degree_ + 1, coefficients_, other.coefficients_, result.coefficients_);
    hipDeviceSynchronize();

    return result;
}

Polynomial& Polynomial::operator=(const Polynomial& other) {
    if (coefficients_ != nullptr) {
        hipFree(coefficients_);
    }
    degree_ = other.degree_;
    hipMalloc((void**)&coefficients_, (degree_ + 1) * sizeof(Fr_t));
    hipMemcpy(coefficients_, other.coefficients_, (degree_ + 1) * sizeof(Fr_t), hipMemcpyDeviceToDevice);
    return *this;
}

__global__ void polyNegKernel(int n, const Fr_t* a, Fr_t* c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = blstrs__scalar__Scalar_sub({0, 0, 0, 0, 0, 0, 0, 0}, a[i]);
    }
}

Polynomial Polynomial::operator-() {
    Polynomial result(degree_);
    polyNegKernel<<<1, degree_ + 1>>>(degree_ + 1, coefficients_, result.coefficients_);
    hipDeviceSynchronize();
    return result;
}

// operator+=   
Polynomial& Polynomial::operator+=(const Polynomial& other)
{
    (*this) = (*this) + other;
    return *this;
}

// operator-=
Polynomial& Polynomial::operator-=(const Polynomial& other)
{
    (*this) = (*this) - other;
    return *this;
}

// operator*=
Polynomial& Polynomial::operator*=(const Polynomial& other)
{
    (*this) = (*this) * other;
    return *this;
}

__global__ void polyEvalKernel(int deg, const Fr_t* coefs, Fr_t x, Fr_t* result_ptr) {
    Fr_t pow = {1, 0, 0, 0, 0, 0, 0, 0};
    *result_ptr = blstrs__scalar__Scalar_ZERO;
    for (int i = 0; i <= deg; ++ i)
    {
        *result_ptr = blstrs__scalar__Scalar_add(*result_ptr, blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(coefs[i], pow)));
        pow = blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(pow, x));
    }
}

Fr_t Polynomial::operator()(const Fr_t& x)
{
    Fr_t* result_ptr;
    hipMalloc((void**)&result_ptr, sizeof(Fr_t));
    polyEvalKernel<<<1, 1>>>(degree_, coefficients_, x, result_ptr);
    hipDeviceSynchronize();
    Fr_t result;
    hipMemcpy(&result, result_ptr, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(result_ptr);
    return result;
}

int Polynomial::getDegree() const {
    return degree_;
}

void Polynomial::setCoefficients(int degree, Fr_t* coefficients){
    if (coefficients_ != nullptr) {
        hipFree(coefficients_);
    }
    degree_ = degree;
    hipMalloc((void**)&coefficients_, (degree_ + 1) * sizeof(Fr_t));
    hipMemcpy(coefficients_, coefficients, (degree_ + 1) * sizeof(Fr_t), hipMemcpyHostToDevice);
}

std::ostream& operator<<(std::ostream& os, const Polynomial& poly)
{
    Fr_t* coefficients = new Fr_t[poly.degree_ + 1];
    hipMemcpy(coefficients, poly.coefficients_, (poly.degree_ + 1) * sizeof(Fr_t), hipMemcpyDeviceToHost);
    for (int i = 0; i <= poly.degree_; i++) {
        os << coefficients[i] << " ";
    }
    delete[] coefficients;
    return os;
}

__global__ void eqPolyKernel(Fr_t u, Fr_t* coefs)
{
    coefs[0] = blstrs__scalar__Scalar_sub({1, 0, 0, 0, 0, 0, 0, 0}, u);
    coefs[1] = blstrs__scalar__Scalar_sub(blstrs__scalar__Scalar_double(u), {1, 0, 0, 0, 0, 0, 0, 0});
}

Polynomial Polynomial::eq(const Fr_t& u){
    Polynomial eq(1);
    eqPolyKernel<<<1, 1>>>(u, eq.coefficients_);
    hipDeviceSynchronize();
    return eq;
}

__global__ void eqEvalKernel(Fr_t u, Fr_t v, Fr_t* eval)
{
    *eval = blstrs__scalar__Scalar_double(blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(u, v)));
    *eval = blstrs__scalar__Scalar_sub(*eval, blstrs__scalar__Scalar_add(u, v));
    *eval = blstrs__scalar__Scalar_add(*eval, {1, 0, 0, 0, 0, 0, 0, 0});
}

Fr_t Polynomial::eq(const Fr_t& u, const Fr_t& v)
{
    Fr_t* eval;
    hipMalloc((void**)&eval, sizeof(Fr_t));
    eqEvalKernel<<<1, 1>>>(u, v, eval);
    hipDeviceSynchronize();
    Fr_t result;
    hipMemcpy(&result, eval, sizeof(Fr_t), hipMemcpyDeviceToHost);
    hipFree(eval);
    return result;
}

// dummy