#include "commitment.cuh"

Commitment Commitment::random(uint size)
{
    Commitment out(size, G1Jacobian_generator);
    out *= FrTensor::random(size);
    return out; 
}

// KERNEL void com_sum_row_kernel(const G1Jacobian_t* arr, G1Jacobian_t* arr_out, uint m, uint n) {
//     auto row = GET_GLOBAL_ID();
//     if (row < m) {
//         G1Jacobian_t rowSum = arr[row * n];
//         for (uint i = 1; i < n; ++ i) {
//             rowSum = blstrs__g1__G1Affine_add(rowSum, arr[row * n + i]);
//         }
//         arr_out[row] = rowSum;
//     }
    
// }

G1TensorJacobian Commitment::commit(const FrTensor& t) const
{
    if (t.size % size != 0) throw std::runtime_error("Commitment::commit - Incompatible dimensions");

    uint m = t.size / size;
    G1TensorJacobian temp = (*this) * t;
    return temp.rowwise_sum(m, size);
}

DEVICE G1Jacobian_t commit_int_dev_func(G1Jacobian_t a, Fr_t s) {
    const int x = scalar_to_int(s);
    G1Jacobian_t out = blstrs__g1__G1Affine_ZERO;
    #pragma unroll
    for (uint i = 0; i < 31; ++ i) {
        if ((x >> i) & 1) out = blstrs__g1__G1Affine_add(out, a);
        a = blstrs__g1__G1Affine_double(a);
    }
    
    if (x < 0) out = blstrs__g1__G1Affine_add(out, G1Jacobian_minus(a));
    return out;
}

KERNEL void commit_int_kernel(const G1Jacobian_t* generators, const Fr_t* scalars, G1Jacobian_t* out, uint n, uint m) {
    const uint gid = GET_GLOBAL_ID();
    if (gid >= m * n) return;
    out[gid] = commit_int_dev_func(generators[gid % n], scalars[gid]);
}

G1TensorJacobian Commitment::commit_int (const FrTensor& t) const{
    if (t.size % size != 0) throw std::runtime_error("Commitment::commit_int - Incompatible dimensions");

    uint m = t.size / size;
    G1TensorJacobian temp(t.size);
    commit_int_kernel<<<(m*size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, temp.gpu_data, size, m);
    hipDeviceSynchronize();
    return temp.rowwise_sum(m, size);
}

G1TensorJacobian Commitment::commit_int_multi(const vector<FrTensor>& ts) const{
    uint num_row = 0;
    for (auto& t : ts) {
        if (t.size % size != 0) throw std::runtime_error("Commitment::commit_int_multi - Incompatible dimensions");
        num_row += t.size / size;
    }

    G1TensorJacobian temp(num_row * size);
    auto temp_start = temp.gpu_data;
    for (auto& t: ts)
    {
        uint m = t.size / size;
        commit_int_kernel<<<(m*size+G1NumThread-1)/G1NumThread,G1NumThread>>>(gpu_data, t.gpu_data, temp_start, size, m);
        hipDeviceSynchronize();
        temp_start += m * size;
    }
    return temp.rowwise_sum(temp.size / size, size);
}

KERNEL void me_open_step(GLOBAL Fr_t* scalars, GLOBAL G1Jacobian_t* generators, Fr_t u, // always assume that scalars and u is in mont form
    GLOBAL Fr_t* new_scalars, GLOBAL G1Jacobian_t* new_generators,
    GLOBAL G1Jacobian_t* temp_out, GLOBAL G1Jacobian_t* temp_out0, GLOBAL G1Jacobian_t* temp_out1, 
    uint old_size, uint new_size)
{
    const uint gid = GET_GLOBAL_ID();
    if (gid >= new_size) return;

    uint gid0 = 2 * gid;
    uint gid1 = 2 * gid + 1;

    if (gid1 >= old_size) {
        new_scalars[gid] = blstrs__scalar__Scalar_sub(scalars[gid0], 
            blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(u, scalars[gid0]))
        );
        new_generators[gid] = G1Jacobian_mul(generators[gid0], u);
        temp_out[gid] = G1Jacobian_mul(generators[gid0], scalars[gid0]);
        temp_out0[gid] = blstrs__g1__G1Affine_ZERO;
        temp_out1[gid] = blstrs__g1__G1Affine_ZERO;
        return;
    }


    new_scalars[gid] = blstrs__scalar__Scalar_add(scalars[gid0], blstrs__scalar__Scalar_mont(blstrs__scalar__Scalar_mul(u, blstrs__scalar__Scalar_sub(scalars[gid1], scalars[gid0]))));
    new_generators[gid] = blstrs__g1__G1Affine_add(generators[gid1], G1Jacobian_mul(blstrs__g1__G1Affine_add(generators[gid0], G1Jacobian_minus(generators[gid1])), u));
    temp_out[gid] = blstrs__g1__G1Affine_add(G1Jacobian_mul(generators[gid0], scalars[gid0]), G1Jacobian_mul(generators[gid1], scalars[gid1]));
    temp_out0[gid] = G1Jacobian_mul(generators[gid1], scalars[gid0]);
    temp_out1[gid] = G1Jacobian_mul(generators[gid0], scalars[gid1]);
}

Fr_t Commitment::me_open(const FrTensor& t, const Commitment& generators, vector<Fr_t>::const_iterator begin, vector<Fr_t>::const_iterator end, vector<G1Jacobian_t>& proof)
{
    if (t.size != generators.size) throw std::runtime_error("Commitment::me_open - Incompatible dimensions "+ std::to_string(t.size) + " " + std::to_string(generators.size));
    if (begin >= end)
    {
        proof.push_back(generators(0));
        return t(0);
    }
    uint new_size = (t.size + 1) / 2;
    FrTensor new_scalars(new_size);
    Commitment new_generators(new_size);
    G1TensorJacobian temp(new_size), temp0(new_size), temp1(new_size);
    me_open_step<<<(new_size+G1NumThread-1)/G1NumThread,G1NumThread>>>(t.gpu_data, generators.gpu_data, *begin, 
    new_scalars.gpu_data, new_generators.gpu_data, temp.gpu_data, temp0.gpu_data, temp1.gpu_data, 
    t.size, new_size);
    hipDeviceSynchronize();
    proof.push_back(temp.sum());
    proof.push_back(temp0.sum());
    proof.push_back(temp1.sum());
    return me_open(new_scalars, new_generators, begin + 1, end, proof);
}



Fr_t Commitment::open(const FrTensor& t, const G1TensorJacobian& com, const vector<Fr_t>& u) const
{
    const vector<Fr_t> u_out(u.end() - ceilLog2(com.size), u.end());
    const vector<Fr_t> u_in(u.begin(), u.end() - ceilLog2(com.size));
    auto g_temp = (com.size == 1)? com(0) : com(u_out);
    // if (size != (1 << u_in.size())) throw std::runtime_error("Incompatible dimensions");
    vector<G1Jacobian_t> proof;
    return me_open(t.partial_me(u_out, t.size / com.size), *this, u_in.begin(), u_in.end(), proof);
}

Weight create_weight(string generator_filename, string weight_filename, string com_filename, uint in_dim, uint out_dim) {
    Commitment generator(generator_filename);
    FrTensor weight = FrTensor::from_int_bin(weight_filename);
    G1TensorJacobian com(com_filename);
    return {generator, weight, com, in_dim, out_dim};
}